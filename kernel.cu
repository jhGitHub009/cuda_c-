#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>
#include<iostream>
#include<math.h>
#include<time.h>

//namespace for standard lib.
using namespace std;

//Calculate error value at each position
__global__ void calculate_error_points(float *totalError, double *b, double *m, float *pointX, float *pointY, int num_points)
{
	int i = threadIdx.x;
	for (i; i<num_points; i++)
	{
		totalError[i] = powf((pointY[i] - (*m * pointX[i] + *b)), 2.0);
	}
}
//calculate for distance between line and points
float compute_error_for_line_given_points(double *b, double *m, float *pointX, float *pointY, int num_points)
{
	float pointError[100] = { 0.0, };
	float totalError = 0.0;
	float ret = 0.0;
	float *d_pointX, *d_pointY, *d_pointError;
	double *d_b, *d_m;
	
	//alocate GPU memory
	hipMalloc(&d_pointX, 100 * sizeof(float));
	hipMalloc(&d_pointY, 100 * sizeof(float));
	hipMalloc(&d_pointError, 100 * sizeof(float));
	hipMalloc(&d_b, sizeof(double));
	hipMalloc(&d_m, sizeof(double));

	//copy CPU data to GPU memory
	hipMemcpy(d_pointX, pointX, 100 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_pointY, pointY, 100 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_pointError, pointError, 100 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_m, m, sizeof(double), hipMemcpyHostToDevice);


	//caluation error between line and points
	calculate_error_points << <1, 100 >> > (d_pointError, d_b, d_m, d_pointX, d_pointY, num_points);
	//getback data from GPU to CPU
	hipMemcpy(pointError, d_pointError, 100 * sizeof(float), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < 100; i++) {
		totalError += pointError[i];
	}
	ret = (totalError / float(num_points));

	//free memory
	hipFree(d_pointX);
	hipFree(d_pointY);
	hipFree(d_pointError);
	hipFree(d_b);
	hipFree(d_m);

	return ret;
}
//calculation gradient for updating weight - for backpropagation.
__global__ void calculate_step_gradient(float *d_b_gradient, float *d_m_gradient, float *d_N, double *d_new_b, double *d_new_m, double *d_b_current, double *d_m_current, float *d_pointX, float *d_pointY, float *d_learningRate, int num_points) {

	int i = threadIdx.x;

	for (int i; i < num_points; i++) {
		*d_b_gradient += -(2 / (*d_N)) * (d_pointY[i] - ((*d_m_current * d_pointX[i]) + *d_b_current));
		*d_m_gradient += -(2 / (*d_N)) * d_pointX[i] * (d_pointY[i] - ((*d_m_current * d_pointX[i]) + *d_b_current));
		*d_new_b = *d_b_current - (*d_learningRate * (*d_b_gradient));
		*d_new_m = *d_m_current - (*d_learningRate * (*d_m_gradient));
	}
}

void step_gradient(double *new_b, double *new_m, double *b_current, double *m_current, float *pointX, float *pointY, float learningRate, int num_points) {
	float b_gradient = 0.0;
	float m_gradient = 0.0;
	float N = float(num_points);

	float *d_pointX, *d_pointY;
	float *d_b_gradient, *d_m_gradient, *d_N, *d_learningRate;
	double *d_new_b, *d_new_m, *d_b_current, *d_m_current;

	//alocate
	hipMalloc(&d_pointX, 100 * sizeof(float));
	hipMalloc(&d_pointY, 100 * sizeof(float));
	hipMalloc(&d_b_gradient, sizeof(float));
	hipMalloc(&d_m_gradient, sizeof(float));
	hipMalloc(&d_N, sizeof(float));
	hipMalloc(&d_new_b, sizeof(double));
	hipMalloc(&d_new_m, sizeof(double));
	hipMalloc(&d_b_current, sizeof(double));
	hipMalloc(&d_m_current, sizeof(double));
	hipMalloc(&d_learningRate, sizeof(float));
	//copy
	hipMemcpy(d_pointX, pointX, 100 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_pointY, pointY, 100 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b_gradient, &b_gradient, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_m_gradient, &m_gradient, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_N, &N, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_new_b, new_b, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_new_m, new_m, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b_current, b_current, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_m_current, m_current, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_learningRate, &learningRate, sizeof(float), hipMemcpyHostToDevice);

	calculate_step_gradient << <1, 100 >> > (d_b_gradient, d_m_gradient, d_N, d_new_b, d_new_m, d_b_current, d_m_current, d_pointX, d_pointY, d_learningRate, num_points);

	hipMemcpy(new_b, d_new_b, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(new_m, d_new_m, sizeof(double), hipMemcpyDeviceToHost);
	//printf("new_b: %f, new_m: %f\n", *new_b, *new_m);
}
//new bais and weight update - backpropagation
void gradient_descent_runner(double *b, double *m, float *pointX, float *pointY, float starting_b, float starting_m, float learning_rate, int num_iterations)
{
	*b = starting_b;
	*m = starting_m;
	double new_b = 0;
	double new_m = 0;

	for (int i = 0; i < num_iterations; i++)
	{
		step_gradient(&new_b, &new_m, b, m, pointX, pointY, learning_rate, 100);
		*b = new_b;
		*m = new_m;
		//printf("d_b_current: %f, d_m_current: %f\n", new_b, new_m);
	}
}
//main event
int main()
{
	//check for time interval
	clock_t begin, end;
	//start time
	begin = clock();
	
	float f1, f2;
	float pointX[100], pointY[100];
	FILE *fp;
	// read CVS
	fp = fopen("C:/Users/user/Desktop/data.csv", "r");
	int i = 0;
	while (fscanf(fp, "%g,%g\n", &f1, &f2) == 2)
	{
		pointX[i] = f1;
		pointY[i] = f2;
		//printf("%g, %g\n", f1, f2);
		i++;
	}

	float learning_rate = 0.0001;
	double initial_b = 0;
	double initial_m = 0;
	int num_iterations = 1000000;
	float error = 0;
	double b = 0.0;
	double m = 0.0;
	//calculate first total error
	error = compute_error_for_line_given_points(&initial_b, &initial_m, pointX, pointY, 100);
	printf("Starting gradient descent at b = %f, m = %f, error = %f\n", initial_b, initial_m, error);
	printf("Running...\n");
	//calculation and update weight and bias.
	gradient_descent_runner(&b, &m, pointX, pointY, initial_b, initial_m, learning_rate, num_iterations);
	//calculate error after backpropagation
 	error = compute_error_for_line_given_points(&b, &m, pointX, pointY, 100);
	printf("After %d iterations b = %f, m = %f, error = %f\n", num_iterations, b, m, error);
	
	//end time
	end = clock();
	printf("GPU time inverval : %d msec\n",(end - begin));
	return 0;
}